#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	printf("Hello World - Block : %d - Thread : %d - Global Thread ID : %d\n" , 
		blockIdx.x, threadIdx.x,idx);
}

int main()
{
	int num_threads, num_blocks;

	printf("Enter number of blocks and threads per block !\n");
	scanf("%d%d",&num_blocks,&num_threads);

	kernel<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();
	printf("Hello World !!\n");

}