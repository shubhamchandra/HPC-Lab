#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 32 // dim of matrix


//Fattened matrix multiplication . Kernel does not support x,y addressing
__global__ void mat_multiply(int* d_mat1, int* d_mat2, int* d_mat3, int width)
{
	int k,sum=0;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;

	if(row<width && col<width)
	{
		for(k=0;k<width;k++)
		{
			sum += d_mat1[row*width+k] * d_mat2[k*width+col];
		}
		d_mat3[row*width+col] = sum;
	}

}

int main()
{
	int i,j;
	int SIZE = N*N;
	int BYTES = SIZE*sizeof(int);

	// declare device and host variables
	int h_mat1[N][N] , h_mat2[N][N] , h_mat3[N][N];
	int *d_mat1, *d_mat2, *d_mat3;

	// allocate memory on the device
	hipMalloc((void**)&d_mat1,BYTES);
	hipMalloc((void**)&d_mat2,BYTES);
	hipMalloc((void**)&d_mat3,BYTES);

	// generate matrix on host
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			h_mat1[i][j] = 1;
			h_mat2[i][j] = 1;
			h_mat3[i][j] = 0;
		}
	}

	dim3 dimGrid(1,1);
	dim3 dimBlock(N,N);

	// move variables from host to device
	hipMemcpy(d_mat1,h_mat1,BYTES,hipMemcpyHostToDevice);
	hipMemcpy(d_mat2,h_mat2,BYTES,hipMemcpyHostToDevice);

	// lauch kernel
	mat_multiply<<<dimGrid,dimBlock>>>(d_mat1,d_mat2,d_mat3,N);
	hipDeviceSynchronize();

	// move result back to main memory
	hipMemcpy(h_mat3,d_mat3,BYTES,hipMemcpyDeviceToHost);

	//print result
	for(i=0;i<N;i++)
	{
		for(j=0;j<N;j++)
		{
			printf("%d ",h_mat3[i][j]);
		}
		printf("\n");
	}

}