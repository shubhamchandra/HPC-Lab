#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int* d_vec1, int* d_vec2, int* d_vec3)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	d_vec3[idx] = d_vec2[idx] + d_vec1[idx];
}

int main()
{
	int i ;

	int num_blocks = 1000;
	int num_threads = 512;

	int SIZE = num_threads*num_blocks;
	int BYTES = SIZE * sizeof(int);

	// declare device and host variables
	int h_vec1[SIZE],h_vec2[SIZE],h_vec3[SIZE];
	int *d_vec1, *d_vec2, *d_vec3;

	// allocate memory on the device
	hipMalloc((void**)&d_vec1,BYTES);
	hipMalloc((void**)&d_vec2,BYTES);
	hipMalloc((void**)&d_vec3,BYTES);

	// generate array on host
	for(i=0;i<SIZE;i++)
	{
		h_vec1[i] = rand()%20;
		h_vec2[i] = rand()%20;
		h_vec3[i] = 0;
	}

	// move variables from host to device
	hipMemcpy(d_vec1,h_vec1,BYTES,hipMemcpyHostToDevice);
	hipMemcpy(d_vec2,h_vec2,BYTES,hipMemcpyHostToDevice);

	// lauch kernel
	add<<<num_blocks,num_threads>>>(d_vec1,d_vec2,d_vec3);

	// move result back to main memory
	hipMemcpy(h_vec3,d_vec3,BYTES,hipMemcpyDeviceToHost);

	//print result
	for(i=0;i<SIZE;i++)
		printf("%d ", h_vec3[i]);

}